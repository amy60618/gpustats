#include "hip/hip_runtime.h"
#ifndef _INCLUDED_MVNPDF
#define _INCLUDED_MVNPDF

#ifdef __cplusplus
extern "C" {
#endif

#include "mvnpdf.h"

typedef struct {
  int data_per_block;
  int params_per_block;
} TuningInfo;


int smem_size() {
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  return deviceProp.sharedMemPerBlock;
}

int max_block_threads() {
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  return deviceProp.maxThreadsPerBlock;
}

int compute_shmem(PMatrix* data, PMatrix* params, int nparams, int ndata) {
  // to hold specified about of data, parameters, and results
  int result_space = nparams * ndata;
  int param_space = params->stride * nparams;
  int data_space = data->cols * ndata;

  return sizeof(float) * (result_space + param_space + data_space);
}

int next_pow2(int k, int pow2) {
  // next highest power of two
  while (k <= pow2 / 2) pow2 /= 2;
  return pow2;
}

int get_boxes(int n, int box_size) {
  // how many boxes of size box_size are needed to hold n things
  return (n + box_size - 1) / box_size;
}

void get_tuned_layout(TuningInfo* info, PMatrix* data, PMatrix* params) {
  // query the device for smem / max # of threads
  int max_smem = smem_size();
  int max_threads = max_block_threads();

  // at most max_block_params sets of density parameters per block
  // for low-dimensional data, better to do more?
  int max_block_params = 16;
  int params_per = max_block_params;
  if (params->rows < max_block_params)
    params_per = next_pow2(params->rows, max_block_params);

  // hide your kids, hide your wife (auto-tuning the GPU)
  int data_per;
  while (1) {
    data_per = max_threads / params_per;
      while (compute_shmem(data, params, params_per, data_per) > max_smem) {
        if (data_per == 0)
          break;
        data_per /=2;
      }

      // can't fit max_block_params sets of parameters into the shared memory,
      // uh oh
      if (data_per == 0) {
        params_per /= 2;

        // start over the tuning
        continue;
      }
      else break;
  }

  info->data_per_block = data_per;
  info->params_per_block = params_per;
}

void inline h_to_d(float* h_ptr, float* d_ptr, size_t n){
  hipError_t error;
  CATCH_ERR(hipMemcpy(d_ptr, h_ptr, n * sizeof(float), hipMemcpyHostToDevice));
}

void inline d_to_h(float* d_ptr, float* h_ptr, size_t n){
  hipError_t error;
  CATCH_ERR(hipMemcpy(h_ptr, d_ptr, n * sizeof(float), hipMemcpyDeviceToHost));
}

__device__ int next_multiple(int k, int mult) {
  if (k % mult)
    return k + (mult - k % mult);
  else
    return k;
}

__device__ float compute_pdf(float* data, float* params, int iD) {
  unsigned int LOGDET_OFFSET = iD * (iD + 3) / 2;
  float* mean = params;
  float* sigma = params + iD;
  float mult = params[LOGDET_OFFSET];
  float logdet = params[LOGDET_OFFSET + 1];

  float discrim = 0;
  float sum;

  for (int i = 0; i < iD; ++i)
  {
    sum = 0;
    for(int j=0; j <= i; j++) {
      sum += *sigma++ * (data[j] - mean[j]);
    }
    discrim += sum * sum;
  }
  return log(mult) - 0.5 * (discrim + logdet + LOG_2_PI * (float) iD);
}

__global__ void mvnpdf_k(const PMatrix data, const PMatrix params, float* output) {

  // coalesce data into shared memory in chunks
  const int num_threads = blockDim.x * blockDim.y;

  // threads in row-major order, better perf
  int thidx = threadIdx.x;
  int thidy = threadIdx.y;

  int tid = thidy * blockDim.x + thidx;

  // now compute your own pdf
  // need to coalesce back into global memory?
  int obs_num = blockDim.x * blockIdx.x + thidx;
  int param_index = blockIdx.y * blockDim.y + thidy;

  // set up shared data
  extern __shared__ float sData[];

  float* sh_params = sData; // store parameters
  float* sh_data = sh_params + blockDim.y * params.stride; // store data
  float* sh_result = sh_data + blockDim.x * data.cols; // store pdfs

  for (int chunk = 0; chunk < data.cols; chunk += blockDim.y)
  {
    if (chunk + thidy < data.cols) {
      sh_data[thidx * data.cols + chunk + thidy] = \
        data.buf[data.stride * obs_num + chunk + thidy];
    }
  }

  for (int chunk = 0; chunk < params.stride; chunk += blockDim.x)
  {
    if (chunk + thidx < params.stride)
      sh_params[thidy * params.stride + chunk + thidx] = \
        params.buf[params.stride * param_index + chunk + thidx];
  }

  // int idx;
  // const int data_start = blockDim.x * blockIdx.x * data.cols;
  // const int data_total = data.rows * data.cols;

  // for (int chunk = data_start;
  //       chunk < data_start + blockDim.x * data.stride;
  //       chunk += num_threads)
  // {
  //    idx = chunk + tid;
  //    if (idx < data_total)
  //      sh_data[idx - data_start] = data.buf[idx];
  // }

  // const int params_start = blockDim.y * blockIdx.y * params.stride;
  // const int params_total = params.rows * params.stride;
  // for (int chunk = params_start;
  //       chunk < params_start + blockDim.y * params.stride;
  //       chunk += num_threads)
  // {
  //    idx = chunk + tid;
  //    if (idx < params_total)
  //      sh_params[idx - params_start] = params.buf[idx];
  // }

  __syncthreads();

  int sh_idx = thidy * blockDim.x + thidx;
  if (obs_num < data.rows & param_index < params.rows) {
    float d = compute_pdf(sh_data + thidx * data.cols,
                          sh_params + thidy * params.stride,
                          data.cols);
    sh_result[sh_idx] = d;
  }
  __syncthreads();

  int result_idx = params.rows * obs_num + param_index;
  if (obs_num < data.rows & param_index < params.rows) {
    output[result_idx] = sh_result[sh_idx];
  }

  // // write out in other order to coalesce
  // // transpose! to get it to coalesce
  // const int result_idx = param_index * data.rows + obs_num;

  // // thread number in column-major order
  // tid = thidx * blockDim.y + thidy;
  // obs_num = blockDim.x * blockIdx.x + tid / blockDim.y;
  // param_index = blockIdx.y * blockDim.y + tid % blockDim.y;
  // const int result_idx = params.rows * obs_num + tid % blockDim.y;

  // if (obs_num < data.rows & param_index < params.rows) {
  //    float d = compute_pdf(sh_data + thidx * data.cols,
  //                          sh_params + thidy * params.stride,
  //                          data.cols);
  //    sh_result[thidx * blockDim.x + thidy] = d;
  // }
  // __syncthreads();

  // // int result_idx = params.rows * obs_num + param_index;
  // int result_idx = (blockIdx.x * blockDim.x * params.rows
  //                    + blockIdx.y * blockDim.y + thidy * params.rows
  //                    + thidx);
  // if (obs_num < data.rows & param_index < params.rows) {
  //    output[result_idx] = sh_result[thidx + thidy * blockDim.y];
  // }
}

hipError_t invoke_mvnpdf(PMatrix data, PMatrix params, float* d_pdf) {
  // Need to automatically tune block / grid layout to maximize shared memory
  // usage and coalescence, reduce wasted threads!
  TuningInfo tune_info;
  get_tuned_layout(&tune_info, &data, &params);

  // Now set up grid layout / block size
  int grid_x = get_boxes(data.rows, tune_info.data_per_block);
  int grid_y = get_boxes(params.rows, tune_info.params_per_block);
  dim3 gridPDF(grid_x, grid_y);

  dim3 blockPDF(tune_info.data_per_block,
                tune_info.params_per_block);

  int sharedMemSize = compute_shmem(&data, &params,
                                    tune_info.params_per_block,
                                    tune_info.data_per_block);

  printf("number params: %d, number data points: %d\n",
         tune_info.params_per_block, tune_info.data_per_block);
  printf("sharedMemSize: %d\n", sharedMemSize);
  printf("block: %d x %d, grid: %d x %d\n", blockPDF.x, blockPDF.y,
         gridPDF.x, gridPDF.y);
  printf("nparams: %d\n", params.rows);

  mvnpdf_k<<<gridPDF,blockPDF,sharedMemSize>>>(data, params, d_pdf);
  return hipSuccess;
}

void mvnpdf2(float* h_data, /** Data-vector; padded */
             float* h_params, /** Density info; already padded */
             float* h_pdf, /** Resultant PDF */
             int data_dim,
             int total_obs,
             int nparams, // multiple sets of parameters
             int param_stride, // with padding
             int data_stride // with padding
  ) {

  float* d_data;
  float* d_params;
  float* d_pdf;
  hipError_t error;

  PMatrix pdata, pparams;
  CATCH_ERR(hipMalloc(&d_pdf, total_obs * nparams * sizeof(float)));
  hipMemset((void*) d_pdf, 1, total_obs * nparams * sizeof(float));

  CATCH_ERR(hipMalloc(&d_data, data_stride * total_obs * sizeof(float)));
  CATCH_ERR(hipMalloc(&d_params, param_stride * nparams * sizeof(float)));

  h_to_d(h_data, d_data, total_obs * data_stride);
  h_to_d(h_params, d_params, nparams * param_stride);

  PMatrix_init(&pdata, d_data, total_obs, data_dim, data_stride);
  PMatrix_init(&pparams, d_params, nparams,
               data_dim * (data_dim + 3) / 2 + 2, param_stride);

  printf("data dim: %d\n", pdata.cols);
  printf("data padded dim: %d\n", pdata.stride);

  invoke_mvnpdf(pdata, pparams, d_pdf);
  d_to_h(d_pdf, h_pdf, total_obs * nparams);

  hipFree(d_data);
  hipFree(d_params);
  hipFree(d_pdf);
}

void cpu_mvnormpdf(float* x, float* density, float * output, int D, int N, int T) {
    int LOGDET_OFFSET = D * (D + 3) / 2;
    int MEAN_CHD_DIM = D * (D + 3) / 2  + 2;
    int PACK_DIM = 16;

    while (MEAN_CHD_DIM > PACK_DIM) {PACK_DIM += 16;}
    int DATA_PADDED_DIM = 8;
    while (D > DATA_PADDED_DIM) {DATA_PADDED_DIM += 8;}

    float* xx = (float*) malloc(D * sizeof(float));
    int obs, component;

    for (obs = 0; obs < N; obs++) {
        for (component = 0; component < T; component++) {
            float discrim;
            float* tData = x + obs * DATA_PADDED_DIM;
            float* tDensityInfo = density + component * PACK_DIM;
            float* tMean = tDensityInfo;            //do we need to unallocate shared/register variables?
            float* tSigma = tDensityInfo + D;
            float  tP = tDensityInfo[LOGDET_OFFSET];
            float  tLogDet = tDensityInfo[LOGDET_OFFSET+1];

            // Do density calculation
            discrim = 0;
            for(int i=0; i<D; i++) {
                float sum = 0;
                for(int j=0; j<=i; j++) {
                  // printf("%d %d %f %f %f\n", i, j, *tSigma, tData[j], tMean[j]);
                  sum += *tSigma * (tData[j] - tMean[j]); // xx[j] is always calculated since j <= i
                  tSigma++;
                }

                discrim += sum * sum;
            }

            float d = log(tP) - 0.5 * (discrim + tLogDet + (LOG_2_PI*(float) D));
            // printf("discrim: %f\n", discrim);
            // printf("tP: %f\n", tP);
            // printf("tLogDet: %f\n", tLogDet);
            // printf("d: %f\n", d);
            // printf("idx: %d\n", obs * T + component);
            output[obs * T + component] = d;
        }
    }
    free(xx);
}


#ifdef __cplusplus
}
#endif

#endif // _INCLUDED_MVNPDF
