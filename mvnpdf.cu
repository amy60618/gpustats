#include "hip/hip_runtime.h"
#ifndef _INCLUDED_MVNPDF
#define _INCLUDED_MVNPDF

#ifdef __cplusplus
extern "C" {
#endif

#include "mvnpdf.h"
#include "cucommon.h"

int compute_shmem(PMatrix* data, PMatrix* params, int nparams, int ndata) {
  // to hold specified about of data, parameters, and results
  int result_space = nparams * ndata;
  int param_space = params->stride * nparams;
  int data_space = data->cols * ndata;

  return sizeof(float) * (result_space + param_space + data_space);
}

void get_tuned_layout(TuningInfo* info, PMatrix* data, PMatrix* params,
                      int max_block_params) {
  // query the device for smem / max # of threads
  int max_smem = smem_size();
  int max_threads = max_block_threads();

  // at most max_block_params sets of density parameters per block
  // for low-dimensional data, better to do more?
  int params_per = max_block_params;
  if (params->rows < max_block_params)
    params_per = next_pow2(params->rows, max_block_params);

  int data_per = max_threads / params_per;
  // at least 16 data points per block
  while (data_per < 16) {
    params_per /= 2;
    data_per *= 2;
  }

  int half_warp = 16;

  // hide your kids, hide your wife (auto-tuning the GPU)
  while (1) {
    while (compute_shmem(data, params, params_per, data_per) > max_smem) {
      if (data_per == 0)
        break;

      if (params_per < half_warp) {
        // no half-warp to protect, maximize data per block
        params_per /= 2;
      }
      else {
        // keep the half-warp
        if (data_per > 1) {
          // TODO: should ask what is the half warp size instead of 16
          if (params_per > 16) {
            params_per /= 2;
            data_per *= 2;
          }
          else {
            // have to do less data
            data_per /= 2;
          }
        }
        else {
          params_per /= 2;
          data_per *= 2;
        }
      }
    }
    // can't fit max_block_params sets of parameters into the shared memory,
    // uh oh
    if (data_per == 0) {
      params_per /= 2;

      // start over the tuning
      continue;
    }
    else break;
  }

  // possible to squeeze more data?
  while (compute_shmem(data, params, params_per, 2 * data_per) <= max_smem)
    if (2 * data_per * params_per <= max_threads)
      data_per *= 2;
    else
      break;

  info->data_per_block = data_per;
  info->params_per_block = params_per;
}

__device__ int d_next_multiple(int k, int mult) {
  if (k % mult)
    return k + (mult - k % mult);
  else
    return k;
}

int next_multiple(int k, int mult) {
  if (k % mult)
    return k + (mult - k % mult);
  else
    return k;
}

__device__ float compute_pdf(float* data, float* params, int dim) {
  unsigned int LOGDET_OFFSET = dim * (dim + 3) / 2;
  float* mean = params;
  float* sigma = params + dim;
  float mult = params[LOGDET_OFFSET];
  float logdet = params[LOGDET_OFFSET + 1];

  float discrim = 0;
  float sum;

  int i, j;
  for (i = 0; i < dim; ++i)
  {
    sum = 0;
    for(j = 0; j <= i; ++j) {
      sum += *sigma++ * (data[j] - mean[j]);
    }
    discrim += sum * sum;
  }
  return log(mult) - 0.5 * (discrim + logdet + LOG_2_PI * (float) dim);
}

__device__ void copy_data(const PMatrix* data, float* sh_data,
                          int thidx, int thidy, int obs_num)
{
  if (obs_num >= data->rows)
    return;

  for (int chunk = 0; chunk < data->cols; chunk += blockDim.y)
  {
    if (chunk + thidy < data->cols) {
      sh_data[thidx * data->cols + chunk + thidy] = \
        data->buf[data->stride * obs_num + chunk + thidy];
    }
  }
  __syncthreads();
}

__device__ void copy_params(const PMatrix* params, float* sh_params,
                            int thidx, int thidy, int param_index)
{
  if (param_index >= params->rows)
    return;

  for (int chunk = 0; chunk < params->stride; chunk += blockDim.x)
  {
    if (chunk + thidx < params->stride)
      sh_params[thidy * params->stride + chunk + thidx] = \
        params->buf[params->stride * param_index + chunk + thidx];
  }
  __syncthreads();
}

__global__ void mvnpdf_k(const PMatrix data, const PMatrix params, float* output) {

  // threads in row-major order, better perf?
  int thidx = threadIdx.x;
  int thidy = threadIdx.y;

  int obs_num = blockDim.x * blockIdx.x + thidx;
  int param_index = blockIdx.y * blockDim.y + thidy;
  int result_idx = params.rows * obs_num + param_index;

  // set up shared data
  extern __shared__ float sData[];

  float* sh_params = sData; // store parameters
  float* sh_data = sh_params + blockDim.y * params.stride; // store data
  float* sh_result = sh_data + blockDim.x * data.cols; // store pdfs

  // coalesce data into shared memory in chunks
  copy_data(&data, sh_data, thidx, thidy, obs_num);
  copy_params(&params, sh_params, thidx, thidy, param_index);

  int sh_idx = thidy * blockDim.x + thidx;
  // allocated enough shared memory so that this will not walk out of bounds
  // no matter what
  sh_result[sh_idx] = compute_pdf(sh_data + thidx * data.cols,
                                  sh_params + thidy * params.stride,
                                  data.cols);
  __syncthreads();

  // does this coalesce?
  if (obs_num < data.rows & param_index < params.rows) {
    output[result_idx] = sh_result[sh_idx];
  }
}


/*
__device__ void _write_results(PMatrix* data, PMatrix* params,
                               float* output, float* sh_result,
                               int thidx, int thidy,
                               int tid)
{
  // write out in other order to coalesce
  // transpose! to get it to coalesce
  const int result_idx = param_index * data.rows + obs_num;

  // thread number in column-major order
  tid = thidx * blockDim.y + thidy;
  obs_num = blockDim.x * blockIdx.x + tid / blockDim.y;
  param_index = blockIdx.y * blockDim.y + tid % blockDim.y;
  const int result_idx = params.rows * obs_num + tid % blockDim.y;

  if (obs_num < data.rows & param_index < params.rows) {
     float d = compute_pdf(sh_data + thidx * data.cols,
                           sh_params + thidy * params.stride,
                           data.cols);
     sh_result[thidx * blockDim.x + thidy] = d;
  }
  __syncthreads();

  // int result_idx = params.rows * obs_num + param_index;
  int result_idx = (blockIdx.x * blockDim.x * params.rows
                     + blockIdx.y * blockDim.y + thidy * params.rows
                     + thidx);
  if (obs_num < data.rows & param_index < params.rows) {
     output[result_idx] = sh_result[thidx + thidy * blockDim.y];
  }
}
*/

int MAX_BLOCK_PARAMS = 64;

hipError_t invoke_mvnpdf(PMatrix data, PMatrix params, float* d_pdf) {
  // Need to automatically tune block / grid layout to maximize shared memory
  // usage and coalescence, reduce wasted threads!
  TuningInfo tune_info;
  get_tuned_layout(&tune_info, &data, &params, MAX_BLOCK_PARAMS);

  // Now set up grid layout / block size
  int grid_x = get_boxes(data.rows, tune_info.data_per_block);
  int grid_y = get_boxes(params.rows, tune_info.params_per_block);
  dim3 gridPDF(grid_x, grid_y);

  dim3 blockPDF(tune_info.data_per_block,
                tune_info.params_per_block);

  int sharedMemSize = compute_shmem(&data, &params,
                                    tune_info.params_per_block,
                                    tune_info.data_per_block);
  printf("number params: %d, number data points: %d\n",
         tune_info.params_per_block, tune_info.data_per_block);
  printf("sharedMemSize: %d\n", sharedMemSize);
  printf("block: %d x %d, grid: %d x %d\n", blockPDF.x, blockPDF.y,
         gridPDF.x, gridPDF.y);
  printf("nparams: %d\n", params.rows);

  mvnpdf_k<<<gridPDF,blockPDF,sharedMemSize>>>(data, params, d_pdf);
  return hipSuccess;
}

void mvnpdf(float* h_data, /** Data-vector; padded */
             float* h_params, /** Density info; already padded */
             float* h_pdf, /** Resultant PDF */
             int data_dim,
             int total_obs,
             int nparams, // multiple sets of parameters
             int param_stride, // with padding
             int data_stride // with padding
  ) {

  float* d_data;
  float* d_params;
  float* d_pdf;
  hipError_t error;

  PMatrix pdata, pparams;
  CATCH_ERR(hipMalloc((void**) &d_pdf, total_obs * nparams * sizeof(float)));
  CATCH_ERR(hipMalloc((void**) &d_data,
                       data_stride * total_obs * sizeof(float)));
  CATCH_ERR(hipMalloc((void**) &d_params,
                       param_stride * nparams * sizeof(float)));

  h_to_d(h_data, d_data, total_obs * data_stride);
  h_to_d(h_params, d_params, nparams * param_stride);

  PMatrix_init(&pdata, d_data, total_obs, data_dim, data_stride);
  PMatrix_init(&pparams, d_params, nparams,
               data_dim * (data_dim + 3) / 2 + 2, param_stride);

  // printf("data dim: %d\n", pdata.cols);
  // printf("data padded dim: %d\n", pdata.stride);

  invoke_mvnpdf(pdata, pparams, d_pdf);
  d_to_h(d_pdf, h_pdf, total_obs * nparams);

  hipFree(d_data);
  hipFree(d_params);
  hipFree(d_pdf);
}

void cpu_mvnormpdf(float* x, float* density, float * output, int D, int N, int T) {
    int LOGDET_OFFSET = D * (D + 3) / 2;
    int MEAN_CHD_DIM = D * (D + 3) / 2  + 2;

    int PACK_DIM = next_multiple(MEAN_CHD_DIM, 16);
    int DATA_PADDED_DIM = next_multiple(D, 8);

    float* xx = (float*) malloc(D * sizeof(float));
    int obs, component;

    for (obs = 0; obs < N; obs++) {
        for (component = 0; component < T; component++) {
            float discrim;
            float* tData = x + obs * DATA_PADDED_DIM;
            float* tDensityInfo = density + component * PACK_DIM;
            float* tMean = tDensityInfo;
            float* tSigma = tDensityInfo + D;
            float  tP = tDensityInfo[LOGDET_OFFSET];
            float  tLogDet = tDensityInfo[LOGDET_OFFSET+1];

            // Do density calculation
            discrim = 0;
            for(int i=0; i<D; i++) {
                float sum = 0;
                for(int j=0; j<=i; j++) {
                  sum += *tSigma * (tData[j] - tMean[j]); // xx[j] is always calculated since j <= i
                  tSigma++;
                }

                discrim += sum * sum;
            }

            float d = log(tP) - 0.5 * (discrim + tLogDet + (LOG_2_PI*(float) D));
            output[obs * T + component] = d;
        }
    }
    free(xx);
}


#ifdef __cplusplus
}
#endif

#endif // _INCLUDED_MVNPDF
