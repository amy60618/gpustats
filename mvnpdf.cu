#include "hip/hip_runtime.h"
#ifndef _INCLUDED_MVNPDF
#define _INCLUDED_MVNPDF

#ifdef __cplusplus
extern "C" {
#endif

#include "mvnpdf.h"

#define BLOCK_SIZE 16
#define BLOCK_TOTAL 256

void inline h_to_d(float* h_ptr, float* d_ptr, int n){
  hipMemcpy(d_ptr, h_ptr, n * sizeof(float), hipMemcpyHostToDevice);
}

void inline d_to_h(float* d_ptr, float* h_ptr, int n){
  hipMemcpy(h_ptr, d_ptr, n * sizeof(float), hipMemcpyDeviceToHost);
}

__device__ int next_multiple(int k, int mult) {
  if (k % mult)
	return k + (mult - k % mult);
  else
	return k;
}

typedef struct {
  REAL* data;
  int rows;
  int pad_columns;
  int columns;
  int stride;
} PMatrix;

/* Thread-Block design:
 * 1 thread per datum*density
 * Block grid(DATA_IN_BLOCK,DENSITIES_IN_BLOCK)
 * DATA_IN_BLOCK = # of datum per block
 * DENSITIES_IN_BLOCK = # of densities per block
 */
#define TWISTED_DENSITY
__global__ void mvNormalPDF(
                    REAL* inData, /** Data-vector; padded */
                    REAL* inDensityInfo, /** Density info; already padded */
                    REAL* outPDF, /** Resultant PDF */
                    int iD,
                    int iN,
                    int iTJ,
                    int isLogScaled
                ) {
    const int thidx = threadIdx.x;
    const int thidy = threadIdx.y;

    const int dataBlockIndex = blockIdx.x * DATA_IN_BLOCK;
    const int datumIndex = dataBlockIndex + thidx;

    const int densityBlockIndex = blockIdx.y * DENSITIES_IN_BLOCK;
    const int densityIndex = densityBlockIndex + thidy;

    #if defined(TWISTED_DENSITY)
        const int pdfIndex = blockIdx.x * DATA_IN_BLOCK * iTJ +
            blockIdx.y * DENSITIES_IN_BLOCK + thidy * iTJ + thidx;
    #else
        const int pdfIndex = datumIndex * iTJ + densityIndex;
    #endif

    extern __shared__ REAL sData[];
    REAL *densityInfo = sData;
    // do this for now, will be more efficient to pass them in as parameters?
    //-------------------------------------------------------
    int LOGDET_OFFSET = iD * (iD + 3) / 2;
    int MEAN_CHD_DIM = iD * (iD + 3) / 2    + 2;
    int PACK_DIM = 16;
    while (MEAN_CHD_DIM > PACK_DIM) {PACK_DIM += 16;}
    int DATA_PADDED_DIM = BASE_DATAPADED_DIM;
    while (iD > DATA_PADDED_DIM) {DATA_PADDED_DIM += BASE_DATAPADED_DIM;}
    //--------------------------------------------------

    const int data_offset = DENSITIES_IN_BLOCK * PACK_DIM;
    REAL *data = &sData[data_offset];

    #if defined(TWISTED_DENSITY)
        REAL *result_trans = &sData[data_offset+DATA_IN_BLOCK * iD];
    #endif

    //Read in data
    for(int chunk = 0; chunk < iD; chunk += DENSITIES_IN_BLOCK)
    if (chunk + thidy < iD ) {
        data[thidx * iD + chunk + thidy] = inData[DATA_PADDED_DIM*datumIndex + chunk + thidy];
    }


    // Read in density info by chunks
    for(int chunk = 0; chunk < PACK_DIM; chunk += DATA_IN_BLOCK) {
        if (chunk + thidx < PACK_DIM) {
            densityInfo[thidy * PACK_DIM + chunk + thidx] = inDensityInfo[PACK_DIM*densityIndex + chunk + thidx];
        }
    }
    __syncthreads();

    // Setup pointers
    REAL* tData = data+thidx*iD;
    REAL* tDensityInfo = densityInfo + thidy * PACK_DIM;


    REAL* tMean = tDensityInfo;         //do we need to unallocate shared/register variables?
    REAL* tSigma = tDensityInfo + iD;
    REAL  tP = tDensityInfo[LOGDET_OFFSET];
    REAL  tLogDet = tDensityInfo[LOGDET_OFFSET+1];

    // Do density calculation
    REAL discrim = 0;
    for(int i=0; i<iD; i++) {
        REAL sum = 0;
        for(int j=0; j<=i; j++) {
            sum += *tSigma++ * (tData[j] - tMean[j]); // xx[j] is always calculated since j <= i
        }
        discrim += sum * sum;
    }
    REAL d;
	REAL mydim = (REAL)iD;
    if (isLogScaled>0) {
	  d = log(tP)-0.5 * (discrim + tLogDet + (LOG_2_PI * mydim));
    } else {
	  d = tP * exp(-0.5 * (discrim + tLogDet + (LOG_2_PI*mydim)));
    }
    #if defined(TWISTED_DENSITY)
        result_trans[thidx * DATA_IN_BLOCK + thidy] = d;
        __syncthreads();
    #endif


    if (datumIndex < iN & densityIndex < iTJ) {
        #if defined(TWISTED_DENSITY)
            outPDF[pdfIndex] = result_trans[thidx + thidy * DENSITIES_IN_BLOCK];
        #else

            outPDF[pdfIndex] = d;
        #endif
    }
}

__device__ float compute_pdf(float* data, float* params, int iD) {
  const int LOGDET_OFFSET = iD * (iD + 3) / 2;
  float* mean = params;
  float* sigma = params + iD;
  float mult = params[LOGDET_OFFSET];
  float logdet = params[LOGDET_OFFSET + 1];

  float discrim = 0;
  float sum;

  for (int i = 0; i < iD; ++i)
  {
   	sum = 0;
   	for(int j=0; j <= i; j++) {
   	  sum += *sigma++ * (data[j] - mean[j]);
   	}
   	discrim += sum * sum;
  }

  return log(mult) - 0.5 * (discrim + logdet + LOG_2_PI * (float) iD);
}

__global__ void mvNormalPDF2(
                    float* glob_data, /** Data-vector; padded */
                    float* glob_params, /** Density info; already padded */
                    float* outPDF, /** Resultant PDF */
                    int iD,
                    int iN
                ) {
  int PACK_DIM = next_multiple(iD * (iD + 3) / 2 + 2, 16);
  int DATA_PADDED_DIM = next_multiple(iD, BASE_DATAPADED_DIM);

  const int block_start = blockIdx.x * blockDim.x * blockDim.y;
  const int data_offset = threadIdx.x * blockDim.x + threadIdx.y;
  const int obs_num = block_start + data_offset;
  const int data_start = obs_num * DATA_PADDED_DIM;
  const int rel_data_start = data_offset * DATA_PADDED_DIM;

  extern __shared__ float sData[];

  float* sh_params = sData;
  float* sh_data = sData + PACK_DIM;

  for (int i = data_start; i < data_start + iD; ++i) {
	sh_data[i - block_start * DATA_PADDED_DIM] = glob_data[data_start + i];
  }

  // read mean, cov, scalar, logdet into shared memory
  // TODO: make sure whole thing read into memory
  if (data_offset < PACK_DIM) {
	sh_params[data_offset] = glob_params[data_offset];
  }
  __syncthreads();

  float density = compute_pdf(sh_data + rel_data_start,
							  sh_params, iD);

  __syncthreads();

  if (obs_num < iN) {
	outPDF[obs_num] = density;
  }
}

hipError_t gpuMvNormalPDF2(
                    float* hData, /** Data-vector; padded */
                    float* hParams, /** Density info; already padded */
                    float* hPDF, /** Resultant PDF */
                    int iD,
                    int iN,
					int PACK_DIM,
					int PADDED_DATA_DIM
                    ) {

  float* dData;
  float* dParams;
  float* dPDF;

  hipMalloc(&dData, PADDED_DATA_DIM * iN * sizeof(float));
  hipMalloc(&dParams, PACK_DIM * sizeof(float));
  hipMalloc(&dPDF, iN * sizeof(float));

  h_to_d(hData, dData, iN * PADDED_DATA_DIM);
  h_to_d(hParams, dParams, PACK_DIM);

  dim3 gridPDF(iN/BLOCK_TOTAL, 1);
  if (iN % BLOCK_TOTAL != 0)
	gridPDF.x += 1;
  dim3 blockPDF(BLOCK_SIZE, BLOCK_SIZE);
  int sharedMemSize = PACK_DIM * SIZE_REAL + PADDED_DATA_DIM * BLOCK_SIZE * BLOCK_SIZE;
  mvNormalPDF2<<<gridPDF,blockPDF,sharedMemSize>>>(dData, dParams, dPDF, iD, iN);

  d_to_h(dPDF, hPDF, iN);

  hipFree(dData);
  hipFree(dParams);
  hipFree(dPDF);

  return hipSuccess;
}


hipError_t gpuMvNormalPDF(
                    REAL* hData, /** Data-vector; padded */
                    REAL* hParams, /** Density info; already padded */
                    REAL* hPDF, /** Resultant PDF */
                    int iD,
                    int iN,
                    int iTJ,
					int PACK_DIM,
					int DIM
                    ) {

  float* dData;
  float* dParams;
  float* dPDF;

  hipMalloc(&dData, DIM * iN * sizeof(float));
  hipMalloc(&dParams, PACK_DIM * sizeof(float));
  hipMalloc(&dPDF, iN * sizeof(float));

  h_to_d(hData, dData, iN);
  h_to_d(hParams, dParams, PACK_DIM);

  dim3 gridPDF(iN/DATA_IN_BLOCK, iTJ/DENSITIES_IN_BLOCK);
  if (iN % DATA_IN_BLOCK != 0)
	gridPDF.x += 1;
  if (iTJ % DENSITIES_IN_BLOCK != 0)
	gridPDF.y += 1;

  dim3 blockPDF(DATA_IN_BLOCK,DENSITIES_IN_BLOCK);
#if defined(TWISTED_DENSITY)
  int sharedMemSize = (DENSITIES_IN_BLOCK * PACK_DIM + DATA_IN_BLOCK * DIM \
					   + DENSITIES_IN_BLOCK*DATA_IN_BLOCK) * SIZE_REAL;
#else
  int sharedMemSize = (DENSITIES_IN_BLOCK * PACK_DIM + DATA_IN_BLOCK * DIM) * SIZE_REAL;
#endif
#if defined(LOGPDF)
  mvNormalPDF<<<gridPDF,blockPDF,sharedMemSize>>>(dData, dParams, dPDF,iD, iN, iTJ,1);
#else
  mvNormalPDF<<<gridPDF,blockPDF,sharedMemSize>>>(dData, dParams, dPDF, iD, iN, iTJ,0);
#endif

  d_to_h(dPDF, hPDF, iN);

  hipFree(dData);
  hipFree(dParams);
  hipFree(dPDF);

    return hipSuccess;
}

void cpu_mvnormpdf(float* x, float* density, float * output, int D, int N, int T) {
    int LOGDET_OFFSET = D * (D + 3) / 2;
	int MEAN_CHD_DIM = D * (D + 3) / 2	+ 2;
	int PACK_DIM = 16;

	while (MEAN_CHD_DIM > PACK_DIM) {PACK_DIM += 16;}
	int DATA_PADDED_DIM = 8;
	while (D > DATA_PADDED_DIM) {DATA_PADDED_DIM += 8;}

    float* xx = (float*) malloc(D * sizeof(float));
    float mydim = (float) D;
    int data,component;

    for (data = 0; data < N; data++) {
        for (component = 0; component < T; component++) {
            float discrim;
            float* tData = x + data * DATA_PADDED_DIM;
            float* tDensityInfo = density + component * PACK_DIM;

            float* tMean = tDensityInfo;			//do we need to unallocate shared/register variables?
            float* tSigma = tDensityInfo + D;
            float  tP = tDensityInfo[LOGDET_OFFSET];
            float  tLogDet = tDensityInfo[LOGDET_OFFSET+1];

            // Do density calculation
            discrim = 0;
            for(int i=0; i<D; i++) {
                float sum = 0;
                for(int j=0; j<=i; j++) {
                    sum += *tSigma++ * (tData[j] - tMean[j]); // xx[j] is always calculated since j <= i
                }
                discrim += sum * sum;
            }

            float d = log(tP) - 0.5 * (discrim + tLogDet + (LOG_2_PI*mydim));
			// printf("discrim: %f\n", discrim);
			// printf("tP: %f\n", tP);
			// printf("tLogDet: %f\n", tLogDet);
			// printf("d: %f\n", d);
			// printf("idx: %d\n", data * T + component);
            output[data * T + component] = d;
        }
    }
	free(xx);
}


#ifdef __cplusplus
}
#endif

#endif // _INCLUDED_MVNPDF
