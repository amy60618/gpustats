#include "hip/hip_runtime.h"
/*
  Block layout : npmfs x nhelpers
  Grid layout : K x 1
  K is the smallest number such that K * npmfs >= pmf_rows
 */

__global__ void
k_%(name)s(float* g_pmf, /** Precomputed pmf */
		   float* g_urand, /** Precomputed random number */
		   float* g_output, /** Resultant choice */
		   int pmf_rows,
		   int pmf_cols,
		   int pmf_stride
  ) {
  // blockDim.x = number of pmfs sampled from in this block
  // blockDim.y = number of helper threads per pmf
  unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
  unsigned int thidx = threadIdx.x;
  unsigned int npmfs = blockDim.x;

  // Make block size flexible ...
  extern __shared__ float shared_data[];

  float* sh_pmf = shared_data; // npmfs * pmf_stride floats
  float* sh_work = sh_pmf + npmfs * pmf_stride; // nmpfs floats

  // Move pmf data into shared memory
  copy_chunks(g_pmf + npmfs * pmf_stride * blockIdx.x,
			  sh_pmf, tid,
			  min(npmfs,
				  pmf_rows - npmfs * blockIdx.x) * pmf_stride);
  __syncthreads();

  // move uniform random draws into shared memory
  copy_chunks(g_urand + npmfs * blockIdx.x,
			  sh_work, tid,
			  min(npmfs, pmf_rows - npmfs * blockIdx.x));
  __syncthreads();

  // done copying, now move pointer to start of pmf for this row of threads
  sh_pmf = sh_pmf + thidx * pmf_stride;

  // compute normalizing constant using atomic operators?

  // for(int chunk = 0; chunk + thidx < pmf_cols; chunk += blockDim.x) {
  // 	atomic_add(sh_work + thidy, sh_pmf[chunk + thidx]);
  // }

  if (threadIdx.y == 0 && thidx < pmf_rows - npmfs * blockIdx.x) {
	float norm_const = 0;
  	for (int i = 0; i < pmf_cols; ++i) {
  	  norm_const += sh_pmf[i];
  	}

	float draw = sh_work[thidx];

	// replace with scaled cumulative pdf
	sh_pmf[0] /= norm_const;
	sh_work[thidx] = 0;
	if (sh_pmf[0] < draw) {
	  for(int i = 1; i < pmf_cols; i++) {
		sh_pmf[i] = sh_pmf[i-1] + sh_pmf[i] / norm_const;
		if (sh_pmf[i] >= draw) {
		  sh_work[thidx] = i;
		  break;
		}
	  }
	}
  }
  __syncthreads();

  // this is now coalesced
  unsigned int result_id = blockIdx.x * npmfs + tid;
  if (result_id < pmf_rows && tid < npmfs)
    g_output[result_id] = sh_work[tid];

  return;
}
