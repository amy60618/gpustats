#include "hip/hip_runtime.h"
__global__ void
k_%(name)s(float* in_measure, /** Precomputed measure */
		   float* in_random, /** Precomputed random number */
		   int* out_component, /** Resultant choice */
		   int iN, int iT) {

  const int sample_density_block = blockDim.x;
  const int sample_block = blockDim.y;
  const int thidx = threadIdx.x;
  const int thidy = threadIdx.y;
  const int datumIndex = blockIdx.x * sample_block  + thidy;
  const int pdfIndex = datumIndex * iT;
  const int tid = thidy*sample_density_block + thidx;

  // Make block size flexible ...
  extern __shared__ float shared_data[];
  float* measure = shared_data; // sample_block by sample_density_block
  float* sum = measure + sample_block*sample_density_block;
  float* work = sum + sample_block;

  // use 'work' in multiple places to save on memory
  if (thidx == 0) {
    sum[thidy] = 0;
    work[thidy] = -10000;
  }

  //get the max values
  for(int chunk = 0; chunk < iT; chunk += sample_density_block) {
    measure[thidy*sample_block + thidx] = in_measure[pdfIndex + chunk + thidx];
    __syncthreads();

    if (thidx == 0) {
      for(int i=0; i<sample_density_block; i++) {
    if(chunk + i < iT){
      float dcurrent = measure[thidy*sample_block + i];
      if (dcurrent > work[thidy]) {
        work[thidy] = dcurrent;
      }
    }
      }
    }
    __syncthreads();
  }


  //get scaled cummulative pdfs
  for(int chunk = 0; chunk < iT; chunk += sample_density_block) {

    measure[thidy*sample_block + thidx] = in_measure[pdfIndex + chunk + thidx];

    __syncthreads();

    if (thidx == 0) {
      for(int i=0; i<sample_density_block; i++) {
    if (chunk + i < iT){
      //rescale and exp()
      sum[thidy] += expf(measure[thidy*sample_block + i] - work[thidy]);
      measure[thidy*sample_block + i] = sum[thidy];
    }
      }
    }

    if (chunk + thidx < iT)
      in_measure[pdfIndex + chunk + thidx] = measure[thidy*sample_block + thidx];

    __syncthreads();
  }

  if (thidx == 0){
    work[thidy] = 0;
  }

  float* randomNumber = sum;
  const int result_id = blockIdx.x * sample_block + tid;
  if ( tid < sample_block )
    randomNumber[tid] = in_random[result_id] * sum[tid];

  // Find the right bin for the random number ...
  for(int chunk = 0; chunk < iT; chunk += sample_density_block) {

    measure[thidy*sample_block + thidx] = in_measure[pdfIndex + chunk + thidx];
    __syncthreads();

    if (thidx == 0) {

      // storing the index in a float is better because it avoids
      // bank conflicts ...
      for(int i=0; i<sample_density_block; i++) {
    if (chunk + i < iT){
      if (randomNumber[thidy] > measure[thidy*sample_block + i]){
        work[thidy] = i + chunk + 1;
      }
    }
      }
      if ((int) work[thidy] >= iT) {work[thidy] = iT-1;}
    }
  }
  __syncthreads();

  // this is now coalesced
  if (result_id < iN && tid < sample_block)
    out_component[result_id] = (int) work[tid];

}
